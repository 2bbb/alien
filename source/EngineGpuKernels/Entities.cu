﻿#include "Entities.cuh"

#include "Cell.cuh"
#include "Token.cuh"
#include "Particle.cuh"

void Entities::init()
{
    cellPointers.init();
    cells.init();
    tokenPointers.init();
    tokens.init();
    particles.init();
    particlePointers.init();
    dynamicMemory.init();
    dynamicMemory.resize(MAX_STRING_BYTES_PER_CELL);
}

void Entities::free()
{
    cellPointers.free();
    cells.free();
    tokenPointers.free();
    tokens.free();
    particles.free();
    particlePointers.free();
    dynamicMemory.free();
}
