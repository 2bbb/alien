#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <functional>

#include "Base.cuh"
#include "Constants.cuh"
#include "CudaInterface.cuh"
#include "Simulation.cuh"
#include "HostHelper.cuh"

namespace {
	hipStream_t cudaStream;
	SimulationManager *simulationManager;
}

void cudaInit(int2 const &size)
{
	hipStreamCreate(&cudaStream);
	hipSetDevice(0);

	simulationManager = new SimulationManager(size);
	
	auto clusters = simulationManager->data.clustersAC1.getArray(NUM_CLUSTERS);

	for (int i = 0; i < NUM_CLUSTERS; ++i) {
		createCluster(simulationManager->data, &clusters[i], { 0.0f, 0.0f }, { random(1.0f) - 0.5f, random(1.0f) - 0.5f }, random(360.0f), random(0.4f) - 0.2f, 10.0, { rand() % 20 + 1, rand() % 20 + 1 }, size);
		do {
			clusters[i].pos = { random(static_cast<float>(size.x)), random(static_cast<float>(size.y)) };
			centerCluster(&clusters[i]);
			updateAbsPos(&clusters[i]);

		} while (!isClusterPositionFree(&clusters[i], &simulationManager->data));

		drawClusterToMap(&clusters[i], &simulationManager->data);
		updateAngularMass(&clusters[i]);
	}
}


void cudaCalcNextTimestep()
{
	simulationManager->prepareTargetData();

	clusterMovement <<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, cudaStream>>> (simulationManager->data);
	hipDeviceSynchronize();
	particleMovement << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, cudaStream >> > (simulationManager->data);
	hipDeviceSynchronize();
	clearMaps <<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, cudaStream>>> (simulationManager->data);
	hipDeviceSynchronize();

	checkCudaErrors(hipGetLastError());
	
	simulationManager->swapData();
}

DataForAccess cudaGetData()
{
	return simulationManager->getDataForAccess();
}

void cudaDataPtrCorrection()
{
	simulationManager->dataPtrCorrection();
}

void cudaShutdown()
{
	hipDeviceSynchronize();
	hipDeviceReset();

	delete simulationManager;
}

