#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <functional>

#include "CudaBase.cuh"
#include "CudaConstants.cuh"
#include "CudaShared.cuh"
#include "CudaMovement.cuh"
#include "CudaHostHelper.cuh"

namespace {
	hipStream_t cudaStream;
	CudaSimulationManager *cudaSimulationManager;
}

void createCluster(CudaCellCluster* cluster, float2 pos, float2 vel, float angle, float angVel, int2 clusterSize, int2 const &size)
{
	cluster->pos = pos;
	cluster->vel = vel;
	cluster->angle = angle;
	cluster->angularVel = angVel;
	cluster->numCells = clusterSize.x * clusterSize.y;
	cluster->cells = cudaSimulationManager->data.cellsAC1.getArray(clusterSize.x*clusterSize.y);

	for (int x = 0; x < clusterSize.x; ++x) {
		for (int y = 0; y < clusterSize.y; ++y) {
			CudaCell *cell = &cluster->cells[x + y*clusterSize.x];
			cell->relPos = { static_cast<float>(x), static_cast<float>(y) };
			cell->cluster = cluster;
			cell->nextTimestep = nullptr;
			cell->protectionCounter = 0;
			cell->numConnections = 0;
			if (x > 0) {
				cell->connections[cell->numConnections++] = &cluster->cells[x - 1 + y * clusterSize.x];
			}
			if (y > 0) {
				cell->connections[cell->numConnections++] = &cluster->cells[x + (y - 1) * clusterSize.x];
			}
			if (x < clusterSize.x - 1) {
				cell->connections[cell->numConnections++] = &cluster->cells[x + 1 + y * clusterSize.x];
			}
			if (y < clusterSize.y - 1) {
				cell->connections[cell->numConnections++] = &cluster->cells[x + (y + 1) * clusterSize.x];
			}
		}
	}
	centerCluster(cluster);
	updateAbsPos(cluster);
	updateAngularMass(cluster);
}

void cudaInit(int2 const &size)
{
	hipStreamCreate(&cudaStream);
	hipSetDevice(0);

	cudaSimulationManager = new CudaSimulationManager(size);
	
	auto clusters = cudaSimulationManager->data.clustersAC1.getArray(NUM_CLUSTERS);

	for (int i = 0; i < NUM_CLUSTERS; ++i) {
		createCluster(&clusters[i], { 0.0f, 0.0f }, { random(0.5f) - 0.25f, random(0.5f) - 0.25f }, random(360.0f), random(0.2f) - 0.1f, { rand() % 20 + 1, rand() % 20 + 1 }, size);
		do {
			clusters[i].pos = { random(static_cast<float>(size.x)), random(static_cast<float>(size.y)) };
			centerCluster(&clusters[i]);
			updateAbsPos(&clusters[i]);

		} while (!isClusterPositionFree(&clusters[i], &cudaSimulationManager->data));

		drawClusterToMap(&clusters[i], &cudaSimulationManager->data);
		updateAngularMass(&clusters[i]);
	}

}


void cudaCalcNextTimestep()
{

	movement_Kernel <<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, cudaStream>>> (cudaSimulationManager->data);
	hipDeviceSynchronize();
	clearOldMap_Kernel <<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, cudaStream>>> (cudaSimulationManager->data);
	hipDeviceSynchronize();

	checkCudaErrors(hipGetLastError());
	
	cudaSimulationManager->swapData();
	cudaSimulationManager->prepareTargetData();
}

CudaData cudaGetDataRef()
{
	CudaData result;
	result.numClusters = cudaSimulationManager->data.clustersAC1.getNumEntries();
	result.clusters = cudaSimulationManager->data.clustersAC1.getEntireArray();
	return result;
}


void cudaShutdown()
{
	hipDeviceSynchronize();
	hipDeviceReset();

	delete cudaSimulationManager;
}

