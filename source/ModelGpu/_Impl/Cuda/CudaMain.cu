#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <functional>

#include "CudaBase.cuh"
#include "CudaConstants.cuh"
#include "CudaShared.cuh"
#include "CudaDeviceFunctions.cuh"

hipStream_t cudaStream;
CudaData cudaData;

void init_Cuda(int2 size)
{
	hipStreamCreate(&cudaStream);
	hipSetDevice(0);

	cudaData.size = size;
	size_t mapSize = size.x * size.y * sizeof(CellCuda*) * LAYERS;
	hipMallocManaged(&cudaData.map1, mapSize);
	hipMallocManaged(&cudaData.map2, mapSize);
	for (int i = 0; i < size.x * size.y * LAYERS; ++i) {
		cudaData.map1[i] = nullptr;
		cudaData.map1[i] = nullptr;
		cudaData.map2[i] = nullptr;
		cudaData.map2[i] = nullptr;
	}
	int cellsPerCluster = 32;
	cudaData.clustersAC1 = ArrayController<ClusterCuda>(NUM_CLUSTERS * 2);
	cudaData.cellsAC1 = ArrayController<CellCuda>(NUM_CLUSTERS * cellsPerCluster * 2);
	cudaData.clustersAC2 = ArrayController<ClusterCuda>(NUM_CLUSTERS * 2);
	cudaData.cellsAC2 = ArrayController<CellCuda>(NUM_CLUSTERS * cellsPerCluster * 2);

	auto clusters = cudaData.clustersAC1.getArray(NUM_CLUSTERS);
	for (int i = 0; i < NUM_CLUSTERS; ++i) {
		clusters[i].pos = { random(size.x), random(size.y) };
		clusters[i].vel = { random(1.0f) - 0.5f, random(1.0) - 0.5f };
		clusters[i].angle = random(360.0f);
		clusters[i].angularVel = random(10.0f) - 5.0f;
		clusters[i].numCells = cellsPerCluster;

		clusters[i].cells = cudaData.cellsAC1.getArray(cellsPerCluster);
		for (int j = 0; j < cellsPerCluster; ++j) {
			CellCuda *cell = &clusters[i].cells[j];
			cell->relPos = { j - 20.0f, j - 20.0f };
			cell->absPos = clusters[i].pos;
			cell->cluster = &clusters[i];
			cell->nextTimestep = nullptr;
			if (j > 0 && j < cellsPerCluster - 1) {
				cell->numConnections = 2;
				cell->connections[0] = &clusters[i].cells[j - 1];
				cell->connections[1] = &clusters[i].cells[j + 1];
			}
			if (j == 0) {
				cell->numConnections = 1;
				cell->connections[0] = &clusters[i].cells[j + 1];
			}
			if (j == cellsPerCluster - 1) {
				cell->numConnections = 1;
				cell->connections[0] = &clusters[i].cells[j - 1];
			}
		}

	}
}

void calcNextTimestep_Cuda()
{
	movement_Kernel <<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, cudaStream>>> (cudaData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	swap(cudaData.clustersAC1, cudaData.clustersAC2);
	swap(cudaData.cellsAC1, cudaData.cellsAC2);
	cudaData.clustersAC2.reset();
	cudaData.cellsAC2.reset();
}

void getDataRef_Cuda(int& numClusters, ClusterCuda*& clusters)
{
	numClusters = cudaData.clustersAC1.getNumEntries();
	clusters = cudaData.clustersAC1.getEntireArray();
}


void end_Cuda()
{
	hipDeviceSynchronize();
	hipDeviceReset();

	cudaData.cellsAC1.free();
	cudaData.clustersAC1.free();
	cudaData.cellsAC2.free();
	cudaData.clustersAC2.free();
	hipFree(cudaData.map1);
	hipFree(cudaData.map2);
}

