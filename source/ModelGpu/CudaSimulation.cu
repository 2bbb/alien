#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>
#include <iostream>
#include <functional>

#include "ModelBasic/SimulationParameters.h"
#include "Base.cuh"
#include "CudaSimulation.cuh"
#include "CudaConstants.cuh"
#include "CudaSimulationParameters.cuh"
#include "CudaAccessTOs.cuh"
#include "SimulationKernels.cuh"
#include "AccessKernels.cuh"
#include "CleanupKernels.cuh"

#include "SimulationData.cuh"
#include "Map.cuh"

namespace
{
    class CudaInitializer
    {
    public:
        static void init()
        {
            static CudaInitializer instance;
        }

        CudaInitializer()
        {
            hipSetDevice(0);
            std::cout << "[CUDA] initialized" << std::endl;
        }

        ~CudaInitializer()
        {
            hipDeviceReset();
            std::cout << "[CUDA] closed" << std::endl;
        }
    };
}

CudaSimulation::CudaSimulation(int2 const &size, SimulationParameters const& parameters)
{

    CudaInitializer::init();

    setSimulationParameters(parameters);

    hipStreamCreate(&_cudaStream);
    std::cout << "[CUDA] stream created" << std::endl;

    _internalData = new SimulationData();
    _internalData->size = size;
    _internalData->clusterPointers = ArrayController<Cluster*>(MAX_CELLCLUSTERPOINTERS);
    _internalData->clusterPointersTemp = ArrayController<Cluster*>(MAX_CELLCLUSTERPOINTERS);
    _internalData->clusters = ArrayController<Cluster>(MAX_CELLCLUSTERS);
    _internalData->cellPointers = ArrayController<Cell*>(MAX_CELLPOINTERS);
    _internalData->cellPointersTemp = ArrayController<Cell*>(MAX_CELLPOINTERS);
    _internalData->cells = ArrayController<Cell>(MAX_CELLS);
    _internalData->cellsTemp = ArrayController<Cell>(MAX_CELLS);
    _internalData->tokenPointers = ArrayController<Token*>(MAX_TOKENPOINTERS);
    _internalData->tokenPointersTemp = ArrayController<Token*>(MAX_TOKENPOINTERS);
    _internalData->tokens = ArrayController<Token>(MAX_TOKENS);
    _internalData->tokensTemp = ArrayController<Token>(MAX_TOKENS);
    _internalData->particles = ArrayController<Particle>(MAX_PARTICLES);
    _internalData->particlesNew = ArrayController<Particle>(MAX_PARTICLES);
    checkCudaErrors(hipMalloc(&_internalData->cellMap, size.x * size.y * sizeof(Cell*)));
    checkCudaErrors(hipMalloc(&_internalData->particleMap, size.x * size.y * sizeof(Particle*)));

    std::vector<Cell*> hostCellMap(size.x * size.y, 0);
    std::vector<Particle*> hostParticleMap(size.x * size.y, 0);
    checkCudaErrors(hipMemcpy(_internalData->cellMap, hostCellMap.data(), sizeof(Cell*)*size.x*size.y, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_internalData->particleMap, hostParticleMap.data(), sizeof(Cell*)*size.x*size.y, hipMemcpyHostToDevice));
    _internalData->numberGen.init(RANDOM_NUMBER_BLOCK_SIZE);

    _cudaAccessTO = new DataAccessTO();
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numClusters, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numCells, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numParticles, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numTokens, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->clusters, sizeof(ClusterAccessTO)*MAX_CELLCLUSTERS));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->cells, sizeof(CellAccessTO)*MAX_CELLS));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->particles, sizeof(ParticleAccessTO)*MAX_PARTICLES));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->tokens, sizeof(TokenAccessTO)*MAX_TOKENS));
}

CudaSimulation::~CudaSimulation()
{
    _internalData->clusterPointers.free();
    _internalData->clusterPointersTemp.free();
    _internalData->clusters.free();
    _internalData->cellPointers.free();
    _internalData->cellPointersTemp.free();
    _internalData->cells.free();
    _internalData->cellsTemp.free();
    _internalData->tokenPointers.free();
    _internalData->tokenPointersTemp.free();
    _internalData->tokens.free();
    _internalData->tokensTemp.free();
    _internalData->particles.free();
    _internalData->particlesNew.free();

    checkCudaErrors(hipFree(_internalData->cellMap));
    checkCudaErrors(hipFree(_internalData->particleMap));
    _internalData->numberGen.free();

    checkCudaErrors(hipFree(_cudaAccessTO->numClusters));
    checkCudaErrors(hipFree(_cudaAccessTO->numCells));
    checkCudaErrors(hipFree(_cudaAccessTO->numParticles));
    checkCudaErrors(hipFree(_cudaAccessTO->clusters));
    checkCudaErrors(hipFree(_cudaAccessTO->cells));
    checkCudaErrors(hipFree(_cudaAccessTO->particles));
    checkCudaErrors(hipFree(_cudaAccessTO->tokens));

    delete _cudaAccessTO;
    delete _internalData;

    std::cout << "[CUDA] stream closed" << std::endl;
}

void CudaSimulation::calcNextTimestep()
{
    prepareTargetData();

    tokenProcessingStep1 << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    tokenProcessingStep2 << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    clusterProcessingOnOrigDataStep1 << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    clusterProcessingOnOrigDataStep2 << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    clusterProcessingOnOrigDataStep3 << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    clusterProcessingOnCopyData << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData, _internalData->clusterPointers.retrieveNumEntries());
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    particleProcessingOnOrigDataStep1 << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    particleProcessingOnOrigDataStep2 << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    particleProcessingOnCopyData << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    cleanup();

    cleanupMaps << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    swapData();
}

void CudaSimulation::getSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    checkCudaErrors(hipMemset(_cudaAccessTO->numClusters, 0, sizeof(int)));
    checkCudaErrors(hipMemset(_cudaAccessTO->numCells, 0, sizeof(int)));
    checkCudaErrors(hipMemset(_cudaAccessTO->numParticles, 0, sizeof(int)));
    checkCudaErrors(hipMemset(_cudaAccessTO->numTokens, 0, sizeof(int)));

    getSimulationAccessData << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    checkCudaErrors(hipMemcpy(dataTO.numClusters, _cudaAccessTO->numClusters, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numTokens, _cudaAccessTO->numTokens, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.clusters, _cudaAccessTO->clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.particles, _cudaAccessTO->particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.tokens, _cudaAccessTO->tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyDeviceToHost));
}

void CudaSimulation::setSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    prepareTargetData();

    checkCudaErrors(hipMemcpy(_cudaAccessTO->numClusters, dataTO.numClusters, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numCells, dataTO.numCells, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numParticles, dataTO.numParticles, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numTokens, dataTO.numTokens, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->clusters, dataTO.clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->cells, dataTO.cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->particles, dataTO.particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->tokens, dataTO.tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyHostToDevice));

    filterData << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (rectUpperLeft, rectLowerRight, *_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    setSimulationAccessData << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >> > (rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    cleanup();

    swapData();
}

void CudaSimulation::setSimulationParameters(SimulationParameters const & parameters)
{
    CudaSimulationParameters parametersToCopy;
    parametersToCopy.cellMaxDistance = parameters.cellMaxDistance;
    parametersToCopy.cellMinDistance = parameters.cellMinDistance;
    parametersToCopy.cellMinEnergy = parameters.cellMinEnergy;
    parametersToCopy.cellFusionVelocity = parameters.cellFusionVelocity;
    parametersToCopy.cellMaxForce = parameters.cellMaxForce;
    parametersToCopy.cellMaxForceDecayProb = parameters.cellMaxForceDecayProb;
    parametersToCopy.cellTransformationProb = parameters.cellTransformationProb;
    parametersToCopy.cellMass = 1.0f / parameters.cellMass_Reciprocal;
    parametersToCopy.cellMaxToken = parameters.cellMaxToken;
    parametersToCopy.cellMaxTokenBranchNumber = parameters.cellMaxTokenBranchNumber;
    parametersToCopy.cellFunctionComputerMaxInstructions = parameters.cellFunctionComputerMaxInstructions;
    parametersToCopy.cellFunctionComputerCellMemorySize = parameters.cellFunctionComputerCellMemorySize;
    parametersToCopy.tokenMinEnergy = parameters.tokenMinEnergy;
    parametersToCopy.tokenMemorySize = parameters.tokenMemorySize;
    parametersToCopy.radiationProbability = parameters.radiationProb;
    parametersToCopy.radiationExponent = parameters.radiationExponent;
    parametersToCopy.radiationFactor = parameters.radiationFactor;
    parametersToCopy.radiationVelocityMultiplier = parameters.radiationVelocityMultiplier;
    parametersToCopy.radiationVelocityPerturbation = parameters.radiationVelocityPerturbation;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &parametersToCopy, sizeof(CudaSimulationParameters), 0, hipMemcpyHostToDevice));
}

void CudaSimulation::prepareTargetData()
{
    _internalData->particlesNew.reset();
}

void CudaSimulation::swapData()
{
    swap(_internalData->particles, _internalData->particlesNew);
}

void CudaSimulation::cleanup()
{
    _internalData->clusterPointersTemp.reset();

    cleanupClusterPointers<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream >>> (*_internalData);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    swap(_internalData->clusterPointers, _internalData->clusterPointersTemp);
}
