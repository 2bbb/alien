#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>
#include <iostream>
#include <functional>

#include "ModelBasic/SimulationParameters.h"
#include "Base.cuh"
#include "CudaSimulation.cuh"
#include "CudaConstants.h"
#include "ConstantMemory.cuh"
#include "AccessTOs.cuh"
#include "SimulationKernels.cuh"
#include "AccessKernels.cuh"
#include "CleanupKernels.cuh"
#include "MonitorKernels.cuh"
#include "Entities.cuh"
#include "CudaMemoryManager.cuh"
#include "CudaMonitorData.cuh"

#include "SimulationData.cuh"
#include "Map.cuh"


#define GPU_FUNCTION(func, ...) func<<<1, 1>>>(##__VA_ARGS__); \
    hipDeviceSynchronize(); \
    checkCudaErrors(hipGetLastError());

namespace
{
    class CudaInitializer
    {
    public:
        static void init()
        {
            static CudaInitializer instance;
        }

        CudaInitializer()
        {
            hipSetDevice(0);
            std::cout << "[CUDA] initialized" << std::endl;
        }

        ~CudaInitializer()
        {
            hipDeviceReset();
            std::cout << "[CUDA] closed" << std::endl;
        }
    };
}

CudaSimulation::CudaSimulation(
    int2 const& size,
    SimulationParameters const& parameters,
    CudaConstants const& cudaConstants)
{

    CudaInitializer::init();
    CudaMemoryManager::getInstance().reset();

    setSimulationParameters(parameters);
    setCudaConstants(cudaConstants);

    _cudaSimulationData = new SimulationData();
    _cudaAccessTO = new DataAccessTO();
    _cudaMonitorData = new CudaMonitorData();

    auto const memorySizeBefore = CudaMemoryManager::getInstance().getSizeOfAcquiredMemory();

    _cudaSimulationData->init(size, cudaConstants);
    _cudaMonitorData->init();

    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numClusters);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().acquireMemory<ClusterAccessTO>(cudaConstants.MAX_CLUSTERS, _cudaAccessTO->clusters);
    CudaMemoryManager::getInstance().acquireMemory<CellAccessTO>(cudaConstants.MAX_CELLS, _cudaAccessTO->cells);
    CudaMemoryManager::getInstance().acquireMemory<ParticleAccessTO>(cudaConstants.MAX_PARTICLES, _cudaAccessTO->particles);
    CudaMemoryManager::getInstance().acquireMemory<TokenAccessTO>(cudaConstants.MAX_TOKENS, _cudaAccessTO->tokens);

    auto const memorySizeAfter = CudaMemoryManager::getInstance().getSizeOfAcquiredMemory();

    std::cout << "[CUDA] " << (memorySizeAfter - memorySizeBefore) / (1024 * 1024) << "mb memory acquired" << std::endl;
}

CudaSimulation::~CudaSimulation()
{
    _cudaSimulationData->free();
    _cudaMonitorData->free();

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numClusters);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->clusters);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->tokens);

    std::cout << "[CUDA] memory released" << std::endl;

    delete _cudaAccessTO;
    delete _cudaSimulationData;
    delete _cudaMonitorData;

}

void CudaSimulation::calcCudaTimestep()
{
    GPU_FUNCTION(calcSimulationTimestep, *_cudaSimulationData);
}

void CudaSimulation::DEBUG_printNumEntries()
{
    std::cout
        << "Particles: " << _cudaSimulationData->entities.particles.retrieveNumEntries() << "; "
        << "Cells: " << _cudaSimulationData->entities.cells.retrieveNumEntries() << "; "
        << "Clusters: " << _cudaSimulationData->entities.clusters.retrieveNumEntries() << "; "
        << "CellPointers: " << _cudaSimulationData->entities.cellPointers.retrieveNumEntries() << "; "
        << "Tokens: " << _cudaSimulationData->entities.tokens.retrieveNumEntries() << "; "
        << "TokenPointers: " << _cudaSimulationData->entities.tokenPointers.retrieveNumEntries() << "; "
        << std::endl;
}

void CudaSimulation::getSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    GPU_FUNCTION(getSimulationAccessData, rectUpperLeft, rectLowerRight, *_cudaSimulationData, *_cudaAccessTO);

    checkCudaErrors(hipMemcpy(dataTO.numClusters, _cudaAccessTO->numClusters, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numTokens, _cudaAccessTO->numTokens, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.clusters, _cudaAccessTO->clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.particles, _cudaAccessTO->particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.tokens, _cudaAccessTO->tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyDeviceToHost));
}

void CudaSimulation::setSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numClusters, dataTO.numClusters, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numCells, dataTO.numCells, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numParticles, dataTO.numParticles, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numTokens, dataTO.numTokens, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->clusters, dataTO.clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->cells, dataTO.cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->particles, dataTO.particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->tokens, dataTO.tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyHostToDevice));

    GPU_FUNCTION(setSimulationAccessData, rectUpperLeft, rectLowerRight, *_cudaSimulationData, *_cudaAccessTO);
}

MonitorData CudaSimulation::getMonitorData()
{
    GPU_FUNCTION(getCudaMonitorData, *_cudaSimulationData, *_cudaMonitorData);
    return _cudaMonitorData->getMonitorData();
}

void CudaSimulation::setSimulationParameters(SimulationParameters const & parameters)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &parameters, sizeof(SimulationParameters), 0, hipMemcpyHostToDevice));
}

void CudaSimulation::clear()
{
    GPU_FUNCTION(clearData, *_cudaSimulationData);
}

void CudaSimulation::setCudaConstants(CudaConstants const & cudaConstants_)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstants), &cudaConstants_, sizeof(CudaConstants), 0, hipMemcpyHostToDevice));
}
