#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>
#include <iostream>
#include <functional>

#include "ModelBasic/SimulationParameters.h"
#include "Base.cuh"
#include "CudaSimulation.cuh"
#include "CudaConstants.cuh"
#include "CudaSimulationParameters.cuh"
#include "CudaAccessTOs.cuh"
#include "SimulationKernels.cuh"
#include "AccessKernels.cuh"
#include "CleanupKernels.cuh"
#include "Entities.cuh"

#include "SimulationData.cuh"
#include "Map.cuh"


#define GPU_FUNCTION(func, ...) func<<<1, 1>>>(##__VA_ARGS__); \
    hipDeviceSynchronize(); \
    checkCudaErrors(hipGetLastError());

namespace
{
    class CudaInitializer
    {
    public:
        static void init()
        {
            static CudaInitializer instance;
        }

        CudaInitializer()
        {
            hipSetDevice(0);
            std::cout << "[CUDA] initialized" << std::endl;
        }

        ~CudaInitializer()
        {
            hipDeviceReset();
            std::cout << "[CUDA] closed" << std::endl;
        }
    };
}

CudaSimulation::CudaSimulation(int2 const &size, SimulationParameters const& parameters)
{

    CudaInitializer::init();

    setSimulationParameters(parameters);

    _internalData = new SimulationData();
    _internalData->init(size);

    _cudaAccessTO = new DataAccessTO();
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numClusters, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numCells, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numParticles, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numTokens, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->clusters, sizeof(ClusterAccessTO)*MAX_CLUSTERS));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->cells, sizeof(CellAccessTO)*MAX_CELLS));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->particles, sizeof(ParticleAccessTO)*MAX_PARTICLES));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->tokens, sizeof(TokenAccessTO)*MAX_TOKENS));

    std::cout << "[CUDA] memory acquired" << std::endl;
}

CudaSimulation::~CudaSimulation()
{
    _internalData->free();

    checkCudaErrors(hipFree(_cudaAccessTO->numClusters));
    checkCudaErrors(hipFree(_cudaAccessTO->numCells));
    checkCudaErrors(hipFree(_cudaAccessTO->numParticles));
    checkCudaErrors(hipFree(_cudaAccessTO->clusters));
    checkCudaErrors(hipFree(_cudaAccessTO->cells));
    checkCudaErrors(hipFree(_cudaAccessTO->particles));
    checkCudaErrors(hipFree(_cudaAccessTO->tokens));

    std::cout << "[CUDA] freed" << std::endl;

    delete _cudaAccessTO;
    delete _internalData;

}

void CudaSimulation::calcNextTimestep()
{
    GPU_FUNCTION(calcSimulationTimestep, *_internalData);
/*
    std::cout
        << "Clusters: " << _internalData->entities.clusterPointers.retrieveNumEntries() << "; " << _internalData->entities.clusters.retrieveNumEntries() << "  "
        << "Cells: " << _internalData->entities.cellPointers.retrieveNumEntries() << "; " << _internalData->entities.cells.retrieveNumEntries()
        << std::endl;
*/
}

void CudaSimulation::getSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    GPU_FUNCTION(getSimulationAccessData, rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);

    checkCudaErrors(hipMemcpy(dataTO.numClusters, _cudaAccessTO->numClusters, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numTokens, _cudaAccessTO->numTokens, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.clusters, _cudaAccessTO->clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.particles, _cudaAccessTO->particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.tokens, _cudaAccessTO->tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyDeviceToHost));
}

void CudaSimulation::setSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numClusters, dataTO.numClusters, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numCells, dataTO.numCells, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numParticles, dataTO.numParticles, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numTokens, dataTO.numTokens, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->clusters, dataTO.clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->cells, dataTO.cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->particles, dataTO.particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->tokens, dataTO.tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyHostToDevice));

    GPU_FUNCTION(setSimulationAccessData, rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);
}

void CudaSimulation::setSimulationParameters(SimulationParameters const & parameters)
{
    CudaSimulationParameters parametersToCopy;
    parametersToCopy.cellMaxDistance = parameters.cellMaxDistance;
    parametersToCopy.cellMinDistance = parameters.cellMinDistance;
    parametersToCopy.cellMinEnergy = parameters.cellMinEnergy;
    parametersToCopy.cellFusionVelocity = parameters.cellFusionVelocity;
    parametersToCopy.cellMaxForce = parameters.cellMaxForce;
    parametersToCopy.cellMaxForceDecayProb = parameters.cellMaxForceDecayProb;
    parametersToCopy.cellTransformationProb = parameters.cellTransformationProb;
    parametersToCopy.cellMass = 1.0f / parameters.cellMass_Reciprocal;
    parametersToCopy.cellMaxToken = parameters.cellMaxToken;
    parametersToCopy.cellMaxTokenBranchNumber = parameters.cellMaxTokenBranchNumber;
    parametersToCopy.cellFunctionComputerMaxInstructions = parameters.cellFunctionComputerMaxInstructions;
    parametersToCopy.cellFunctionComputerCellMemorySize = parameters.cellFunctionComputerCellMemorySize;
    parametersToCopy.tokenMinEnergy = parameters.tokenMinEnergy;
    parametersToCopy.tokenMemorySize = parameters.tokenMemorySize;
    parametersToCopy.radiationProbability = parameters.radiationProb;
    parametersToCopy.radiationExponent = parameters.radiationExponent;
    parametersToCopy.radiationFactor = parameters.radiationFactor;
    parametersToCopy.radiationVelocityMultiplier = parameters.radiationVelocityMultiplier;
    parametersToCopy.radiationVelocityPerturbation = parameters.radiationVelocityPerturbation;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &parametersToCopy, sizeof(CudaSimulationParameters), 0, hipMemcpyHostToDevice));
}
