#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>
#include <iostream>
#include <functional>

#include "ModelBasic/SimulationParameters.h"
#include "Base.cuh"
#include "CudaSimulation.cuh"
#include "CudaConstants.cuh"
#include "CudaSimulationParameters.cuh"
#include "CudaInterface.cuh"
#include "SimulationFunctions.cuh"
#include "AccessFunctions.cuh"

#include "SimulationData.cuh"
#include "Map.cuh"

namespace
{
	class CudaInitializer
	{
	public:
		static void init()
		{
			static CudaInitializer instance;
		}
		
		CudaInitializer()
		{
			hipSetDevice(0);
			std::cout << "[CUDA] initialized" << std::endl;
		}

		~CudaInitializer()
		{
			hipDeviceReset();
			std::cout << "[CUDA] closed" << std::endl;
		}
	};
}

CudaSimulation::CudaSimulation(int2 const &size, SimulationParameters const& parameters)
{

	CudaInitializer::init();

	setSimulationParameters(parameters);

	hipStreamCreate(&_cudaStream);
	std::cout << "[CUDA] stream created" << std::endl;

	_internalData = new SimulationData();
	_internalData->size = size;
	_internalData->clustersAC1 = ArrayController<Cluster>(MAX_CELLCLUSTERS);
	_internalData->clustersAC2 = ArrayController<Cluster>(MAX_CELLCLUSTERS);
	_internalData->cellsAC1 = ArrayController<Cell>(MAX_CELLS);
	_internalData->cellsAC2 = ArrayController<Cell>(MAX_CELLS);
	_internalData->particlesAC1 = ArrayController<Particle>(MAX_PARTICLES);
	_internalData->particlesAC2 = ArrayController<Particle>(MAX_PARTICLES);
	_internalData->tokensAC1 = ArrayController<Token>(MAX_TOKENS);
	_internalData->tokensAC2 = ArrayController<Token>(MAX_TOKENS);
	checkCudaErrors(hipMalloc(&_internalData->cellMap, size.x * size.y * sizeof(Cell*)));
	checkCudaErrors(hipMalloc(&_internalData->particleMap, size.x * size.y * sizeof(Particle*)));

	std::vector<Cell*> hostCellMap(size.x * size.y, 0);
	std::vector<Particle*> hostParticleMap(size.x * size.y, 0);
	checkCudaErrors(hipMemcpy(_internalData->cellMap, hostCellMap.data(), sizeof(Cell*)*size.x*size.y, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_internalData->particleMap, hostParticleMap.data(), sizeof(Cell*)*size.x*size.y, hipMemcpyHostToDevice));
	_internalData->numberGen.init(RANDOM_NUMBER_BLOCK_SIZE);

	_cudaAccessTO = new DataAccessTO();
	checkCudaErrors(hipMalloc(&_cudaAccessTO->numClusters, sizeof(int)));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->numCells, sizeof(int)));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->numParticles, sizeof(int)));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->numTokens, sizeof(int)));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->clusters, sizeof(ClusterAccessTO)*MAX_CELLCLUSTERS));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->cells, sizeof(CellAccessTO)*MAX_CELLS));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->particles, sizeof(ParticleAccessTO)*MAX_PARTICLES));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->tokens, sizeof(TokenAccessTO)*MAX_TOKENS));
}

CudaSimulation::~CudaSimulation()
{
	_internalData->clustersAC1.free();
	_internalData->clustersAC2.free();
	_internalData->cellsAC1.free();
	_internalData->cellsAC2.free();
	_internalData->particlesAC1.free();
	_internalData->particlesAC2.free();
	_internalData->tokensAC1.free();
	_internalData->tokensAC2.free();

	checkCudaErrors(hipFree(_internalData->cellMap));
	checkCudaErrors(hipFree(_internalData->particleMap));
	_internalData->numberGen.free();

	checkCudaErrors(hipFree(_cudaAccessTO->numClusters));
	checkCudaErrors(hipFree(_cudaAccessTO->numCells));
	checkCudaErrors(hipFree(_cudaAccessTO->numParticles));
	checkCudaErrors(hipFree(_cudaAccessTO->clusters));
	checkCudaErrors(hipFree(_cudaAccessTO->cells));
	checkCudaErrors(hipFree(_cudaAccessTO->particles));
	checkCudaErrors(hipFree(_cudaAccessTO->tokens));

	delete _cudaAccessTO;
	delete _internalData;

	std::cout << "[CUDA] stream closed" << std::endl;
}

void CudaSimulation::calcNextTimestep()
{
	prepareTargetData();

	clusterDynamicsStep1<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	clusterDynamicsStep2<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	clusterReorganizing<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	particleDynamicsStep1<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	particleDynamicsStep2<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	particleReorganizing<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	clearMaps<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	swapData();
}

void CudaSimulation::getSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
	checkCudaErrors(hipMemset(_cudaAccessTO->numClusters, 0, sizeof(int)));
	checkCudaErrors(hipMemset(_cudaAccessTO->numCells, 0, sizeof(int)));
	checkCudaErrors(hipMemset(_cudaAccessTO->numParticles, 0, sizeof(int)));
	checkCudaErrors(hipMemset(_cudaAccessTO->numTokens, 0, sizeof(int)));

	getSimulationAccessData<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>>(rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipMemcpy(dataTO.numClusters, _cudaAccessTO->numClusters, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(dataTO.numTokens, _cudaAccessTO->numTokens, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(dataTO.clusters, _cudaAccessTO->clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(dataTO.particles, _cudaAccessTO->particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(dataTO.tokens, _cudaAccessTO->tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyDeviceToHost));
}

void CudaSimulation::setSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
	prepareTargetData();

	checkCudaErrors(hipMemcpy(_cudaAccessTO->numClusters, dataTO.numClusters, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->numCells, dataTO.numCells, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->numParticles, dataTO.numParticles, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->numTokens, dataTO.numTokens, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->clusters, dataTO.clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->cells, dataTO.cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->particles, dataTO.particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->tokens, dataTO.tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyHostToDevice));

	filterData<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (rectUpperLeft, rectLowerRight, *_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	setSimulationAccessData<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	swapData();
}

void CudaSimulation::setSimulationParameters(SimulationParameters const & parameters)
{
	CudaSimulationParameters parametersToCopy;
	parametersToCopy.cellMaxDistance = parameters.cellMaxDistance;
	parametersToCopy.cellMinDistance = parameters.cellMinDistance;
	parametersToCopy.cellMinEnergy = parameters.cellMinEnergy;
	parametersToCopy.cellFusionVelocity = parameters.cellFusionVelocity;
	parametersToCopy.cellMaxForce = parameters.cellMaxForce;
	parametersToCopy.cellMaxForceDecayProb = parameters.cellMaxForceDecayProb;
	parametersToCopy.cellTransformationProb = parameters.cellTransformationProb;
	parametersToCopy.cellMass = 1.0f / parameters.cellMass_Reciprocal;
	parametersToCopy.cellMaxToken = parameters.cellMaxToken;
	parametersToCopy.cellMaxTokenBranchNumber = parameters.cellMaxTokenBranchNumber;
	parametersToCopy.radiationProbability = parameters.radiationProb;
	parametersToCopy.radiationExponent = parameters.radiationExponent;
	parametersToCopy.radiationFactor = parameters.radiationFactor;
	parametersToCopy.radiationVelocityMultiplier = parameters.radiationVelocityMultiplier;
	parametersToCopy.radiationVelocityPerturbation = parameters.radiationVelocityPerturbation;

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &parametersToCopy, sizeof(CudaSimulationParameters), 0, hipMemcpyHostToDevice));
}

void CudaSimulation::prepareTargetData()
{
	_internalData->clustersAC2.reset();
	_internalData->cellsAC2.reset();
	_internalData->particlesAC2.reset();
	_internalData->tokensAC2.reset();
}

void CudaSimulation::swapData()
{
	swap(_internalData->clustersAC1, _internalData->clustersAC2);
	swap(_internalData->cellsAC1, _internalData->cellsAC2);
	swap(_internalData->particlesAC1, _internalData->particlesAC2);
	swap(_internalData->tokensAC1, _internalData->tokensAC2);
}
