#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>
#include <iostream>
#include <functional>

#include "ModelBasic/SimulationParameters.h"
#include "Base.cuh"
#include "CudaSimulation.cuh"
#include "CudaConstants.cuh"
#include "CudaSimulationParameters.cuh"
#include "CudaAccessTOs.cuh"
#include "SimulationKernels.cuh"
#include "AccessKernels.cuh"
#include "CleanupKernels.cuh"

#include "SimulationData.cuh"
#include "Map.cuh"


#define KERNEL_FUNCTION(numBlocks, numThreadsPerBlock, func, ...) func<<<numBlocks, numThreadsPerBlock>>>(##__VA_ARGS__); \
    hipDeviceSynchronize(); \
    checkCudaErrors(hipGetLastError());

namespace
{
    class CudaInitializer
    {
    public:
        static void init()
        {
            static CudaInitializer instance;
        }

        CudaInitializer()
        {
            hipSetDevice(0);
            std::cout << "[CUDA] initialized" << std::endl;
        }

        ~CudaInitializer()
        {
            hipDeviceReset();
            std::cout << "[CUDA] closed" << std::endl;
        }
    };
}

CudaSimulation::CudaSimulation(int2 const &size, SimulationParameters const& parameters)
{

    CudaInitializer::init();

    setSimulationParameters(parameters);

    _internalData = new SimulationData();
    _internalData->size = size;
    _internalData->clusterPointers = ArrayController<Cluster*>(MAX_CELLCLUSTERPOINTERS);
    _internalData->clusterPointersTemp = ArrayController<Cluster*>(MAX_CELLCLUSTERPOINTERS);
    _internalData->clusters = ArrayController<Cluster>(MAX_CELLCLUSTERS);
    _internalData->cellPointers = ArrayController<Cell*>(MAX_CELLPOINTERS);
    _internalData->cellPointersTemp = ArrayController<Cell*>(MAX_CELLPOINTERS);
    _internalData->cells = ArrayController<Cell>(MAX_CELLS);
    _internalData->cellsTemp = ArrayController<Cell>(MAX_CELLS);
    _internalData->tokenPointers = ArrayController<Token*>(MAX_TOKENPOINTERS);
    _internalData->tokenPointersTemp = ArrayController<Token*>(MAX_TOKENPOINTERS);
    _internalData->tokens = ArrayController<Token>(MAX_TOKENS);
    _internalData->tokensTemp = ArrayController<Token>(MAX_TOKENS);
    _internalData->particles = ArrayController<Particle>(MAX_PARTICLES);
    _internalData->particlesNew = ArrayController<Particle>(MAX_PARTICLES);
    checkCudaErrors(hipMalloc(&_internalData->cellMap, size.x * size.y * sizeof(Cell*)));
    checkCudaErrors(hipMalloc(&_internalData->particleMap, size.x * size.y * sizeof(Particle*)));

    std::vector<Cell*> hostCellMap(size.x * size.y, 0);
    std::vector<Particle*> hostParticleMap(size.x * size.y, 0);
    checkCudaErrors(hipMemcpy(_internalData->cellMap, hostCellMap.data(), sizeof(Cell*)*size.x*size.y, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_internalData->particleMap, hostParticleMap.data(), sizeof(Cell*)*size.x*size.y, hipMemcpyHostToDevice));
    _internalData->numberGen.init(RANDOM_NUMBER_BLOCK_SIZE);

    _cudaAccessTO = new DataAccessTO();
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numClusters, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numCells, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numParticles, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->numTokens, sizeof(int)));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->clusters, sizeof(ClusterAccessTO)*MAX_CELLCLUSTERS));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->cells, sizeof(CellAccessTO)*MAX_CELLS));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->particles, sizeof(ParticleAccessTO)*MAX_PARTICLES));
    checkCudaErrors(hipMalloc(&_cudaAccessTO->tokens, sizeof(TokenAccessTO)*MAX_TOKENS));
}

CudaSimulation::~CudaSimulation()
{
    _internalData->clusterPointers.free();
    _internalData->clusterPointersTemp.free();
    _internalData->clusters.free();
    _internalData->cellPointers.free();
    _internalData->cellPointersTemp.free();
    _internalData->cells.free();
    _internalData->cellsTemp.free();
    _internalData->tokenPointers.free();
    _internalData->tokenPointersTemp.free();
    _internalData->tokens.free();
    _internalData->tokensTemp.free();
    _internalData->particles.free();
    _internalData->particlesNew.free();

    checkCudaErrors(hipFree(_internalData->cellMap));
    checkCudaErrors(hipFree(_internalData->particleMap));
    _internalData->numberGen.free();

    checkCudaErrors(hipFree(_cudaAccessTO->numClusters));
    checkCudaErrors(hipFree(_cudaAccessTO->numCells));
    checkCudaErrors(hipFree(_cudaAccessTO->numParticles));
    checkCudaErrors(hipFree(_cudaAccessTO->clusters));
    checkCudaErrors(hipFree(_cudaAccessTO->cells));
    checkCudaErrors(hipFree(_cudaAccessTO->particles));
    checkCudaErrors(hipFree(_cudaAccessTO->tokens));

    delete _cudaAccessTO;
    delete _internalData;

}

void CudaSimulation::calcNextTimestep()
{
    KERNEL_FUNCTION(1, 1, calcSimulationTimestep, *_internalData);
    swap(_internalData->particles, _internalData->particlesNew);
    swap(_internalData->clusterPointers, _internalData->clusterPointersTemp);
}

void CudaSimulation::getSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    KERNEL_FUNCTION(1, 1, getSimulationAccessData, rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);

    checkCudaErrors(hipMemcpy(dataTO.numClusters, _cudaAccessTO->numClusters, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numTokens, _cudaAccessTO->numTokens, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.clusters, _cudaAccessTO->clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.particles, _cudaAccessTO->particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.tokens, _cudaAccessTO->tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyDeviceToHost));
}

void CudaSimulation::setSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numClusters, dataTO.numClusters, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numCells, dataTO.numCells, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numParticles, dataTO.numParticles, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numTokens, dataTO.numTokens, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->clusters, dataTO.clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->cells, dataTO.cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->particles, dataTO.particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->tokens, dataTO.tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyHostToDevice));

    KERNEL_FUNCTION(1, 1, setSimulationAccessData, rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);
    swap(_internalData->particles, _internalData->particlesNew);
    swap(_internalData->clusterPointers, _internalData->clusterPointersTemp);
}

void CudaSimulation::setSimulationParameters(SimulationParameters const & parameters)
{
    CudaSimulationParameters parametersToCopy;
    parametersToCopy.cellMaxDistance = parameters.cellMaxDistance;
    parametersToCopy.cellMinDistance = parameters.cellMinDistance;
    parametersToCopy.cellMinEnergy = parameters.cellMinEnergy;
    parametersToCopy.cellFusionVelocity = parameters.cellFusionVelocity;
    parametersToCopy.cellMaxForce = parameters.cellMaxForce;
    parametersToCopy.cellMaxForceDecayProb = parameters.cellMaxForceDecayProb;
    parametersToCopy.cellTransformationProb = parameters.cellTransformationProb;
    parametersToCopy.cellMass = 1.0f / parameters.cellMass_Reciprocal;
    parametersToCopy.cellMaxToken = parameters.cellMaxToken;
    parametersToCopy.cellMaxTokenBranchNumber = parameters.cellMaxTokenBranchNumber;
    parametersToCopy.cellFunctionComputerMaxInstructions = parameters.cellFunctionComputerMaxInstructions;
    parametersToCopy.cellFunctionComputerCellMemorySize = parameters.cellFunctionComputerCellMemorySize;
    parametersToCopy.tokenMinEnergy = parameters.tokenMinEnergy;
    parametersToCopy.tokenMemorySize = parameters.tokenMemorySize;
    parametersToCopy.radiationProbability = parameters.radiationProb;
    parametersToCopy.radiationExponent = parameters.radiationExponent;
    parametersToCopy.radiationFactor = parameters.radiationFactor;
    parametersToCopy.radiationVelocityMultiplier = parameters.radiationVelocityMultiplier;
    parametersToCopy.radiationVelocityPerturbation = parameters.radiationVelocityPerturbation;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &parametersToCopy, sizeof(CudaSimulationParameters), 0, hipMemcpyHostToDevice));
}
