#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>
#include <iostream>
#include <functional>

#include "Base.cuh"
#include "CudaSimulation.cuh"
#include "CudaConstants.cuh"
#include "SimulationParameters.cuh"
#include "CudaInterface.cuh"
#include "SimulationFunctions.cuh"
#include "AccessFunctions.cuh"

#include "SimulationData.cuh"
#include "Map.cuh"

namespace
{
	class CudaInitializer
	{
	public:
		static void init()
		{
			static CudaInitializer instance;
		}
		
		CudaInitializer()
		{
			hipSetDevice(0);
			std::cout << "[CUDA] initialized" << std::endl;
		}

		~CudaInitializer()
		{
			hipDeviceReset();
			std::cout << "[CUDA] closed" << std::endl;
		}
	};
}

CudaSimulation::CudaSimulation(int2 const &size)
{

	CudaInitializer::init();

	setCudaSimulationParameters();

	hipStreamCreate(&_cudaStream);
	std::cout << "[CUDA] stream created" << std::endl;

	_internalData = new SimulationData();
	_internalData->size = size;
	_internalData->clustersAC1 = ArrayController<Cluster>(MAX_CELLCLUSTERS);
	_internalData->clustersAC2 = ArrayController<Cluster>(MAX_CELLCLUSTERS);
	_internalData->cellsAC1 = ArrayController<Cell>(MAX_CELLS);
	_internalData->cellsAC2 = ArrayController<Cell>(MAX_CELLS);
	_internalData->particlesAC1 = ArrayController<Particle>(MAX_PARTICLES);
	_internalData->particlesAC2 = ArrayController<Particle>(MAX_PARTICLES);
	checkCudaErrors(hipMalloc(&_internalData->cellMap, size.x * size.y * sizeof(Cell*)));
	checkCudaErrors(hipMalloc(&_internalData->particleMap, size.x * size.y * sizeof(Particle*)));

	std::vector<Cell*> hostCellMap(size.x * size.y, 0);
	std::vector<Particle*> hostParticleMap(size.x * size.y, 0);
	checkCudaErrors(hipMemcpy(_internalData->cellMap, hostCellMap.data(), sizeof(Cell*)*size.x*size.y, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_internalData->particleMap, hostParticleMap.data(), sizeof(Cell*)*size.x*size.y, hipMemcpyHostToDevice));
	_internalData->numberGen.init(RANDOM_NUMBER_BLOCK_SIZE);

	_accessTO = new SimulationAccessTO();
	_accessTO->numClusters = new int();
	_accessTO->numCells = new int();
	_accessTO->numParticles = new int();
	_accessTO->clusters = new ClusterAccessTO[MAX_CELLCLUSTERS];
	_accessTO->cells = new CellAccessTO[MAX_CELLS];
	_accessTO->particles = new ParticleAccessTO[MAX_PARTICLES];
	*_accessTO->numClusters = 0;
	*_accessTO->numCells = 0;
	*_accessTO->numParticles = 0;

	_cudaAccessTO = new SimulationAccessTO();
	checkCudaErrors(hipMalloc(&_cudaAccessTO->numClusters, sizeof(int)));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->numCells, sizeof(int)));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->numParticles, sizeof(int)));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->clusters, sizeof(ClusterAccessTO)*MAX_CELLCLUSTERS));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->cells, sizeof(CellAccessTO)*MAX_CELLS));
	checkCudaErrors(hipMalloc(&_cudaAccessTO->particles, sizeof(ParticleAccessTO)*MAX_PARTICLES));
}

CudaSimulation::~CudaSimulation()
{
	_internalData->clustersAC1.free();
	_internalData->clustersAC2.free();
	_internalData->cellsAC1.free();
	_internalData->cellsAC2.free();
	_internalData->particlesAC1.free();
	_internalData->particlesAC2.free();

	checkCudaErrors(hipFree(_internalData->cellMap));
	checkCudaErrors(hipFree(_internalData->particleMap));
	_internalData->numberGen.free();

	checkCudaErrors(hipFree(_cudaAccessTO->numClusters));
	checkCudaErrors(hipFree(_cudaAccessTO->numCells));
	checkCudaErrors(hipFree(_cudaAccessTO->numParticles));
	checkCudaErrors(hipFree(_cudaAccessTO->clusters));
	checkCudaErrors(hipFree(_cudaAccessTO->cells));
	checkCudaErrors(hipFree(_cudaAccessTO->particles));

	delete _accessTO->numClusters;
	delete _accessTO->numCells;
	delete _accessTO->numParticles;
	delete[] _accessTO->clusters;
	delete[] _accessTO->cells;
	delete[] _accessTO->particles;

	delete _accessTO;
	delete _cudaAccessTO;
	delete _internalData;

	std::cout << "[CUDA] stream closed" << std::endl;
}

void CudaSimulation::calcNextTimestep()
{
	prepareTargetData();

	clusterDynamicsStep1<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	clusterDynamicsStep2<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	clusterReorganizing<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	particleDynamicsStep1<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	particleDynamicsStep2<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	particleReorganizing<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	clearMaps<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (*_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	swapData();
}

SimulationAccessTO* CudaSimulation::getSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight)
{
	_rectUpperLeft = rectUpperLeft;
	_rectLowerRight = rectLowerRight;

	checkCudaErrors(hipMemset(_cudaAccessTO->numClusters, 0, sizeof(int)));
	checkCudaErrors(hipMemset(_cudaAccessTO->numCells, 0, sizeof(int)));
	checkCudaErrors(hipMemset(_cudaAccessTO->numParticles, 0, sizeof(int)));

	getSimulationAccessData<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (_rectUpperLeft, _rectLowerRight, *_internalData, *_cudaAccessTO);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipMemcpy(_accessTO->numClusters, _cudaAccessTO->numClusters, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(_accessTO->numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(_accessTO->numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(_accessTO->clusters, _cudaAccessTO->clusters, sizeof(ClusterAccessTO) * (*_accessTO->numClusters), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(_accessTO->cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*_accessTO->numCells), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(_accessTO->particles, _cudaAccessTO->particles, sizeof(ParticleAccessTO) * (*_accessTO->numParticles), hipMemcpyDeviceToHost));

	return _accessTO;
}

void CudaSimulation::updateSimulationData()
{
 	prepareTargetData();

	checkCudaErrors(hipMemcpy(_cudaAccessTO->numClusters, _accessTO->numClusters, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->numCells, _accessTO->numCells, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->numParticles, _accessTO->numParticles, sizeof(int), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->clusters, _accessTO->clusters, sizeof(ClusterAccessTO) * (*_accessTO->numClusters), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->cells, _accessTO->cells, sizeof(CellAccessTO) * (*_accessTO->numCells), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(_cudaAccessTO->particles, _accessTO->particles, sizeof(ParticleAccessTO) * (*_accessTO->numParticles), hipMemcpyHostToDevice));

	filterData<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (_rectUpperLeft, _rectLowerRight, *_internalData);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	setSimulationAccessData<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, _cudaStream>>> (_rectUpperLeft, _rectLowerRight, *_internalData, *_cudaAccessTO);
	hipDeviceSynchronize();
	checkCudaErrors(hipGetLastError());

	swapData();
}

void CudaSimulation::prepareTargetData()
{
	_internalData->clustersAC2.reset();
	_internalData->cellsAC2.reset();
	_internalData->particlesAC2.reset();
}

void CudaSimulation::swapData()
{
	swap(_internalData->clustersAC1, _internalData->clustersAC2);
	swap(_internalData->cellsAC1, _internalData->cellsAC2);
	swap(_internalData->particlesAC1, _internalData->particlesAC2);
}

void CudaSimulation::setCudaSimulationParameters()
{
	SimulationParameters parametersToCopy;
	parametersToCopy.cellMaxDistance = 1.3f;
	parametersToCopy.cellMinDistance = 0.3f;
	parametersToCopy.cellMinEnergy = 50.0f;
	parametersToCopy.cellFusionVelocity = 0.4f;
	parametersToCopy.cellMaxForce = 0.8f;
	parametersToCopy.cellMaxForceDecayProb = 0.2f;
	parametersToCopy.cellTransformationProb = 0.2f;
	parametersToCopy.cellMass = 1.0;
	parametersToCopy.radiationProbability = 0.2f;
	parametersToCopy.radiationExponent = 1.0f;
	parametersToCopy.radiationFactor = 0.0002f;
	parametersToCopy.radiationVelocityMultiplier = 1.0f;
	parametersToCopy.radiationVelocityPerturbation = 0.5f;

	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &parametersToCopy, sizeof(SimulationParameters) , 0, hipMemcpyHostToDevice));
}
