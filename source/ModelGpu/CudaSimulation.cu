#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>
#include <iostream>
#include <functional>

#include "ModelBasic/SimulationParameters.h"
#include "Base.cuh"
#include "CudaSimulation.cuh"
#include "CudaConstants.h"
#include "ConstantMemory.cuh"
#include "AccessTOs.cuh"
#include "SimulationKernels.cuh"
#include "AccessKernels.cuh"
#include "CleanupKernels.cuh"
#include "MonitorKernels.cuh"
#include "RenderingKernels.cuh"
#include "PhysicalActionKernels.cuh"
#include "Entities.cuh"
#include "CudaMemoryManager.cuh"
#include "CudaMonitorData.cuh"

#include "SimulationData.cuh"
#include "Map.cuh"


#define GPU_FUNCTION(func, ...) func<<<1, 1>>>(##__VA_ARGS__); \
    hipDeviceSynchronize(); \
    checkCudaErrors(hipGetLastError());

namespace
{
    class CudaInitializer
    {
    public:
        static void init()
        {
            static CudaInitializer instance;
        }

        CudaInitializer()
        {
            hipSetDevice(0);
            std::cout << "[CUDA] initialized" << std::endl;
        }

        ~CudaInitializer()
        {
            hipDeviceReset();
            std::cout << "[CUDA] closed" << std::endl;
        }
    };
}

CudaSimulation::CudaSimulation(
    int2 const& size,
    int timestep,
    SimulationParameters const& parameters,
    CudaConstants const& cudaConstants)
{

    CudaInitializer::init();
    CudaMemoryManager::getInstance().reset();

    setSimulationParameters(parameters);
    setCudaConstants(cudaConstants);

    _cudaSimulationData = new SimulationData();
    _cudaAccessTO = new DataAccessTO();
    _cudaMonitorData = new CudaMonitorData();

    auto const memorySizeBefore = CudaMemoryManager::getInstance().getSizeOfAcquiredMemory();

    _cudaSimulationData->init(size, cudaConstants, timestep);
    _cudaMonitorData->init();

    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numClusters);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numStringBytes);
    CudaMemoryManager::getInstance().acquireMemory<ClusterAccessTO>(cudaConstants.MAX_CLUSTERS, _cudaAccessTO->clusters);
    CudaMemoryManager::getInstance().acquireMemory<CellAccessTO>(cudaConstants.MAX_CELLS, _cudaAccessTO->cells);
    CudaMemoryManager::getInstance().acquireMemory<ParticleAccessTO>(cudaConstants.MAX_PARTICLES, _cudaAccessTO->particles);
    CudaMemoryManager::getInstance().acquireMemory<TokenAccessTO>(cudaConstants.MAX_TOKENS, _cudaAccessTO->tokens);
    CudaMemoryManager::getInstance().acquireMemory<char>(cudaConstants.METADATA_DYNAMIC_MEMORY_SIZE, _cudaAccessTO->stringBytes);

    auto const memorySizeAfter = CudaMemoryManager::getInstance().getSizeOfAcquiredMemory();

    std::cout << "[CUDA] " << (memorySizeAfter - memorySizeBefore) / (1024 * 1024) << "mb memory acquired" << std::endl;
}

CudaSimulation::~CudaSimulation()
{
    _cudaSimulationData->free();
    _cudaMonitorData->free();

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numClusters);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numStringBytes);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->clusters);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->tokens);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->stringBytes);

    std::cout << "[CUDA] memory released" << std::endl;

    delete _cudaAccessTO;
    delete _cudaSimulationData;
    delete _cudaMonitorData;

}

void CudaSimulation::calcCudaTimestep()
{
    GPU_FUNCTION(calcSimulationTimestep, *_cudaSimulationData);
    ++_cudaSimulationData->timestep;
}

void CudaSimulation::DEBUG_printNumEntries()
{
    std::cout
        << "Particles: " << _cudaSimulationData->entities.particles.retrieveNumEntries() << "; "
        << "Cells: " << _cudaSimulationData->entities.cells.retrieveNumEntries() << "; "
        << "Clusters: " << _cudaSimulationData->entities.clusters.retrieveNumEntries() << "; "
        << "CellPointers: " << _cudaSimulationData->entities.cellPointers.retrieveNumEntries() << "; "
        << "Tokens: " << _cudaSimulationData->entities.tokens.retrieveNumEntries() << "; "
        << "TokenPointers: " << _cudaSimulationData->entities.tokenPointers.retrieveNumEntries() << "; "
        << std::endl;
}

void CudaSimulation::getSimulationImage(int2 const & rectUpperLeft, int2 const & rectLowerRight, unsigned char* imageData)
{
    int width = rectLowerRight.x - rectUpperLeft.x + 1;
    int height = rectLowerRight.y - rectUpperLeft.y + 1;
    int numPixels = width * height;

    GPU_FUNCTION(drawImage, rectUpperLeft, rectLowerRight, *_cudaSimulationData);
    checkCudaErrors(hipMemcpy(
        imageData, _cudaSimulationData->finalImageData, sizeof(unsigned int) * numPixels, hipMemcpyDeviceToHost));
}

void CudaSimulation::getSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    GPU_FUNCTION(getSimulationAccessData, rectUpperLeft, rectLowerRight, *_cudaSimulationData, *_cudaAccessTO);

    checkCudaErrors(hipMemcpy(dataTO.numClusters, _cudaAccessTO->numClusters, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numTokens, _cudaAccessTO->numTokens, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numStringBytes, _cudaAccessTO->numStringBytes, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.clusters, _cudaAccessTO->clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.particles, _cudaAccessTO->particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.tokens, _cudaAccessTO->tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.stringBytes, _cudaAccessTO->stringBytes, sizeof(char) * (*dataTO.numStringBytes), hipMemcpyDeviceToHost));
}

void CudaSimulation::setSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numClusters, dataTO.numClusters, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numCells, dataTO.numCells, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numParticles, dataTO.numParticles, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numTokens, dataTO.numTokens, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numStringBytes, dataTO.numStringBytes, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->clusters, dataTO.clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->cells, dataTO.cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->particles, dataTO.particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->tokens, dataTO.tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->stringBytes, dataTO.stringBytes, sizeof(char) * (*dataTO.numStringBytes), hipMemcpyHostToDevice));

    GPU_FUNCTION(setSimulationAccessData, rectUpperLeft, rectLowerRight, *_cudaSimulationData, *_cudaAccessTO);
}

void CudaSimulation::applyForce(ApplyForceData const& applyData)
{
    CudaApplyForceData cudaApplyData{ applyData.startPos, applyData.endPos, applyData.force, applyData.onlyRotation };
    GPU_FUNCTION(cudaApplyForce, cudaApplyData, *_cudaSimulationData);
}

MonitorData CudaSimulation::getMonitorData()
{
    GPU_FUNCTION(getCudaMonitorData, *_cudaSimulationData, *_cudaMonitorData);
    return _cudaMonitorData->getMonitorData();
}

int CudaSimulation::getTimestep() const
{
    return _cudaSimulationData->timestep;
}

void CudaSimulation::setTimestep(int timestep)
{
    _cudaSimulationData->timestep = timestep;
}

void CudaSimulation::setSimulationParameters(SimulationParameters const & parameters)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(
        cudaSimulationParameters), &parameters, sizeof(SimulationParameters), 0, hipMemcpyHostToDevice));
}

void CudaSimulation::setExecutionParameters(ExecutionParameters const & parameters)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(
        cudaExecutionParameters),
        &parameters,
        sizeof(ExecutionParameters),
        0,
        hipMemcpyHostToDevice));
}

void CudaSimulation::clear()
{
    GPU_FUNCTION(clearData, *_cudaSimulationData);
}

namespace
{
    void calcImageBlurFactors(int* imageBlurFactors)
    {
        imageBlurFactors[0] = 300;
        imageBlurFactors[1] = 40;
        imageBlurFactors[2] = 7;
        imageBlurFactors[3] = 7;
        imageBlurFactors[4] = 7;
        imageBlurFactors[5] = 7;

        int sum = 0;
        int2 relPos;
        for (relPos.x = -5; relPos.x <= 5; ++relPos.x) {
            for (relPos.y = -5; relPos.y <= 5; ++relPos.y) {
                auto r = Math::length(toFloat2(relPos));
                if (r <= 5 + FP_PRECISION) {
                    sum += imageBlurFactors[floorInt(r)];
                }
            }
        }
        imageBlurFactors[6] = sum - 400;
    }
}

void CudaSimulation::setCudaConstants(CudaConstants const & cudaConstants_)
{
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaConstants), &cudaConstants_, sizeof(CudaConstants), 0, hipMemcpyHostToDevice));

    int imageBlurFactors[7];
    calcImageBlurFactors(imageBlurFactors);
    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaImageBlurFactors), &imageBlurFactors, sizeof(int) * 7, 0, hipMemcpyHostToDevice));
}
