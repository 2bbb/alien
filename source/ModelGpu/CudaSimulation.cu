#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>
#include <iostream>
#include <functional>

#include "ModelBasic/SimulationParameters.h"
#include "Base.cuh"
#include "CudaSimulation.cuh"
#include "CudaConstants.cuh"
#include "CudaSimulationParameters.cuh"
#include "CudaAccessTOs.cuh"
#include "SimulationKernels.cuh"
#include "AccessKernels.cuh"
#include "CleanupKernels.cuh"
#include "Entities.cuh"
#include "CudaMemoryManager.cuh"

#include "SimulationData.cuh"
#include "Map.cuh"


#define GPU_FUNCTION(func, ...) func<<<1, 1>>>(##__VA_ARGS__); \
    hipDeviceSynchronize(); \
    checkCudaErrors(hipGetLastError());

namespace
{
    class CudaInitializer
    {
    public:
        static void init()
        {
            static CudaInitializer instance;
        }

        CudaInitializer()
        {
            hipSetDevice(0);
            std::cout << "[CUDA] initialized" << std::endl;
        }

        ~CudaInitializer()
        {
            hipDeviceReset();
            std::cout << "[CUDA] closed" << std::endl;
        }
    };
}

CudaSimulation::CudaSimulation(int2 const &size, SimulationParameters const& parameters)
{

    CudaInitializer::init();
    CudaMemoryManager::getInstance().reset();

    setSimulationParameters(parameters);

    _internalData = new SimulationData();
    _cudaAccessTO = new DataAccessTO();

    _internalData->init(size);

    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numClusters);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().acquireMemory<int>(1, _cudaAccessTO->numTokens);
    CudaMemoryManager::getInstance().acquireMemory<ClusterAccessTO>(MAX_CLUSTERS, _cudaAccessTO->clusters);
    CudaMemoryManager::getInstance().acquireMemory<CellAccessTO>(MAX_CELLS, _cudaAccessTO->cells);
    CudaMemoryManager::getInstance().acquireMemory<ParticleAccessTO>(MAX_PARTICLES, _cudaAccessTO->particles);
    CudaMemoryManager::getInstance().acquireMemory<TokenAccessTO>(MAX_TOKENS, _cudaAccessTO->tokens);

    std::cout << "[CUDA] " << CudaMemoryManager::getInstance().getSizeOfAcquiredMemory() / (1024 * 1024) << "mb memory acquired" << std::endl;
}

CudaSimulation::~CudaSimulation()
{
    _internalData->free();

    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numClusters);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numCells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->numParticles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->clusters);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->cells);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->particles);
    CudaMemoryManager::getInstance().freeMemory(_cudaAccessTO->tokens);

    std::cout << "[CUDA] freed" << std::endl;

    delete _cudaAccessTO;
    delete _internalData;

}

void CudaSimulation::calcNextTimestep()
{
    GPU_FUNCTION(calcSimulationTimestep, *_internalData);
/*
    std::cout
    << "Particles: " << _internalData->entities.particlePointers.retrieveNumEntries() << "; " << _internalData->entities.particles.retrieveNumEntries()
    << std::endl;
*/
}

void CudaSimulation::getSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    GPU_FUNCTION(getSimulationAccessData, rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);

    checkCudaErrors(hipMemcpy(dataTO.numClusters, _cudaAccessTO->numClusters, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numCells, _cudaAccessTO->numCells, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numParticles, _cudaAccessTO->numParticles, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.numTokens, _cudaAccessTO->numTokens, sizeof(int), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.clusters, _cudaAccessTO->clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.cells, _cudaAccessTO->cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.particles, _cudaAccessTO->particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(dataTO.tokens, _cudaAccessTO->tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyDeviceToHost));
}

void CudaSimulation::setSimulationData(int2 const& rectUpperLeft, int2 const& rectLowerRight, DataAccessTO const& dataTO)
{
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numClusters, dataTO.numClusters, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numCells, dataTO.numCells, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numParticles, dataTO.numParticles, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->numTokens, dataTO.numTokens, sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->clusters, dataTO.clusters, sizeof(ClusterAccessTO) * (*dataTO.numClusters), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->cells, dataTO.cells, sizeof(CellAccessTO) * (*dataTO.numCells), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->particles, dataTO.particles, sizeof(ParticleAccessTO) * (*dataTO.numParticles), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(_cudaAccessTO->tokens, dataTO.tokens, sizeof(TokenAccessTO) * (*dataTO.numTokens), hipMemcpyHostToDevice));

    GPU_FUNCTION(setSimulationAccessData, rectUpperLeft, rectLowerRight, *_internalData, *_cudaAccessTO);
}

void CudaSimulation::setSimulationParameters(SimulationParameters const & parameters)
{
    CudaSimulationParameters parametersToCopy;
    parametersToCopy.cellMaxDistance = parameters.cellMaxDistance;
    parametersToCopy.cellMinDistance = parameters.cellMinDistance;
    parametersToCopy.cellMinEnergy = parameters.cellMinEnergy;
    parametersToCopy.cellFusionVelocity = parameters.cellFusionVelocity;
    parametersToCopy.cellMaxForce = parameters.cellMaxForce;
    parametersToCopy.cellMaxForceDecayProb = parameters.cellMaxForceDecayProb;
    parametersToCopy.cellTransformationProb = parameters.cellTransformationProb;
    parametersToCopy.cellMass = 1.0f / parameters.cellMass_Reciprocal;
    parametersToCopy.cellMaxToken = parameters.cellMaxToken;
    parametersToCopy.cellMaxTokenBranchNumber = parameters.cellMaxTokenBranchNumber;
    parametersToCopy.cellFunctionComputerMaxInstructions = parameters.cellFunctionComputerMaxInstructions;
    parametersToCopy.cellFunctionComputerCellMemorySize = parameters.cellFunctionComputerCellMemorySize;
    parametersToCopy.tokenMinEnergy = parameters.tokenMinEnergy;
    parametersToCopy.tokenMemorySize = parameters.tokenMemorySize;
    parametersToCopy.radiationProbability = parameters.radiationProb;
    parametersToCopy.radiationExponent = parameters.radiationExponent;
    parametersToCopy.radiationFactor = parameters.radiationFactor;
    parametersToCopy.radiationVelocityMultiplier = parameters.radiationVelocityMultiplier;
    parametersToCopy.radiationVelocityPerturbation = parameters.radiationVelocityPerturbation;

    checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(cudaSimulationParameters), &parametersToCopy, sizeof(CudaSimulationParameters), 0, hipMemcpyHostToDevice));
}
