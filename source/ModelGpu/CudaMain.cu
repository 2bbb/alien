#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <functional>

#include "Base.cuh"
#include "Constants.cuh"
#include "CudaInterface.cuh"
#include "Simulation.cuh"
#include "SimulationDataManager.cuh"

namespace {
	hipStream_t cudaStream;
	SimulationDataManager *simulationManager;
}

void cudaInit(int2 const &size)
{
	hipStreamCreate(&cudaStream);
	hipSetDevice(0);

	simulationManager = new SimulationDataManager(size);
	
	auto clusters = simulationManager->data.clustersAC1.getArray(NUM_CLUSTERS);

	for (int i = 0; i < NUM_CLUSTERS; ++i) {
		simulationManager->createCluster(simulationManager->data, &clusters[i], { 0.0f, 0.0f }, { random(1.0f) - 0.5f, random(1.0f) - 0.5f }, random(360.0f), random(0.4f) - 0.2f, 100.0, { rand() % 20 + 1, rand() % 20 + 1 }, size);
		do {
			clusters[i].pos = { random(static_cast<float>(size.x)), random(static_cast<float>(size.y)) };
			simulationManager->centerCluster(&clusters[i]);
			simulationManager->updateAbsPos(&clusters[i]);

		} while (!simulationManager->isClusterPositionFree(&clusters[i], &simulationManager->data));

		simulationManager->drawClusterToMap(&clusters[i], &simulationManager->data);
		simulationManager->updateAngularMass(&clusters[i]);
	}
}


void cudaCalcNextTimestep()
{
	simulationManager->prepareTargetData();

	clusterMovement <<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, cudaStream>>> (simulationManager->data);
	hipDeviceSynchronize();
	particleMovement << <NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, cudaStream >> > (simulationManager->data);
	hipDeviceSynchronize();
	clearMaps <<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK, 0, cudaStream>>> (simulationManager->data);
	hipDeviceSynchronize();

	checkCudaErrors(hipGetLastError());
	
	simulationManager->swapData();
}

SimulationDataForAccess cudaGetData()
{
	return simulationManager->getDataForAccess();
}

void cudaShutdown()
{
	hipDeviceSynchronize();
	delete simulationManager;
	hipDeviceReset();

}

