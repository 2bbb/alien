#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>

#include <stdio.h>
#include <functional>

#include "Base.cuh"
#include "Constants.cuh"
#include "CudaInterface.cuh"
#include "Simulation.cuh"
#include "SimulationDataManager.cuh"

