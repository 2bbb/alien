#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>
#include <list>

#include <stdio.h>
#include <functional>

#include "Base.cuh"
#include "Constants.cuh"
#include "CudaInterface.cuh"
#include "Simulation.cuh"
#include "SimulationDataManager.cuh"

//TODO: create class
namespace {
	int instances = 0;
	hipStream_t cudaStream;
	SimulationDataManager *simulationManager;
}

void cudaInit(int2 const &size)
{
	if (instances++ == 0) {
		hipStreamCreate(&cudaStream);
		hipSetDevice(0);
		std::cout << "CUDA stream initialized" << std::endl;
	}

	simulationManager = new SimulationDataManager(size);
}


void cudaCalcNextTimestep()
{
	simulationManager->calcNextTimestep(cudaStream);
}

SimulationDataForAccess cudaGetData()
{
	return simulationManager->getDataForAccess();
}

void cudaSetData(SimulationDataForAccess const& access)
{
	simulationManager->setDataForAccess(access);
}

void cudaShutdown()
{
	hipDeviceSynchronize();
	delete simulationManager;

	if (--instances == 0) {
		hipDeviceReset();

		std::cout << "CUDA stream closed" << std::endl;
	}
}

