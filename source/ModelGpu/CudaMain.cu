#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <functional>

#include "Base.cuh"
#include "Constants.cuh"
#include "CudaInterface.cuh"
#include "Simulation.cuh"
#include "SimulationDataManager.cuh"

namespace {
	hipStream_t cudaStream;
	SimulationDataManager *simulationManager;
}

void cudaInit(int2 const &size)
{
	hipStreamCreate(&cudaStream);
	hipSetDevice(0);

	std::cout << "CUDA stream initialized" << std::endl;

	simulationManager = new SimulationDataManager(size);
}


void cudaCalcNextTimestep()
{
	simulationManager->calcNextTimestep(cudaStream);
}

SimulationDataForAccess cudaGetData()
{
	return simulationManager->getDataForAccess();
}

void cudaSetData(SimulationDataForAccess const& access)
{
	simulationManager->setDataForAccess(access);
}

void cudaShutdown()
{
	hipDeviceSynchronize();
	delete simulationManager;
	hipDeviceReset();

	std::cout << "CUDA stream closed" << std::endl;
}

